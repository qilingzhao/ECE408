#include <iostream>

int main() {
    
    int devCount;
    hipGetDeviceCount(&devCount);
    std::cout << "device count is " << devCount << std::endl;

    hipDeviceProp_t devProp;
    for (uint i = 0; i < devCount; i++) {
        hipGetDeviceProperties(&devProp, i);
        std::cout << "devProp.maxThreadsPerBlock: " << devProp.maxThreadsPerBlock << std::endl;
        std::cout << "the number of SMs(devProp.multiProcessorCount): " << devProp.multiProcessorCount << std::endl;
        std::cout << "clockRate: " << devProp.clockRate << std::endl;
        std::cout << "Maximum number of threads in a block(maxThreadsDim): x: " << devProp.maxThreadsDim[0] <<
                 ", y: " << devProp.maxThreadsDim[1] << 
                 ", z: " << devProp.maxThreadsDim[2] << std::endl;
        std::cout << "Maxium number of blocks in a grid(maxGridSize): x: " << devProp.maxGridSize[0] <<
                ", y: " << devProp.maxGridSize[1] << 
                ", z: " << devProp.maxGridSize[2] << std::endl;
        std::cout << "devProp.regsPerBlock/Grid: " << devProp.regsPerBlock << std::endl;
        std::cout << "devProp.warpSize: " << devProp.warpSize << std::endl;
    }

    return 0;
}
// The output of Tesla T4
// device count is 1
// devProp.maxThreadsPerBlock: 1024
// the number of SMs(devProp.multiProcessorCount): 40
// clockRate: 1590000
// Maximum number of threads in a block(maxThreadsDim): x: 1024, y: 1024, z: 64
// Maxium number of blocks in a grid(maxGridSize): x: 2147483647, y: 65535, z: 65535
// devProp.regsPerBlock/Grid: 65536
// devProp.warpSize: 32
